#include "hip/hip_runtime.h"
/*
This program will numerically compute the integral of
                  4/(1+x*x)
from 0 to 1.  The value of this integral is pi -- which
is great since it gives us an easy way to check the answer.
The is the original sequential program.  It uses the timer
from the OpenMP runtime library
History: Written by Tim Mattson, 11/99.
*/

#include <stdlib.h>
#include "omp_repair.h"
#include "book.h"

const long num_steps = 100000000;
const int nbBlocks = 32;
const int nbThreadPerBlock = 1024;


__global__ void kernel(double step, double * GPU) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x, i;
    double result = 0.0, x;

    for (i = tid; i <= num_steps ; i += nbThreadPerBlock * nbBlocks) {
        x = (i-0.5)*step;
        result = result + 4.0/(1.0+x*x);
    }
    
    // Affectation du resultat d'un seul thread
    GPU[tid] = result;
}

int main ()
{
    double pi, start_time, run_time, step, final = 0, * dev_Sum, * sum;
    
    sum = (double*) malloc((nbThreadPerBlock * nbBlocks) * sizeof(double));

    step = 1.0/(double) num_steps;
    
    start_time = omp_get_wtime();
    
    HANDLE_ERROR(hipMalloc((void**)&dev_Sum, sizeof(double) * (nbBlocks * nbThreadPerBlock)));
    kernel<<<nbBlocks, nbThreadPerBlock>>>(step, dev_Sum);
    HANDLE_ERROR(hipMemcpy(sum, dev_Sum, sizeof(double) * (nbBlocks * nbThreadPerBlock), hipMemcpyDeviceToHost));

    for (int i = 0; i < nbBlocks * nbThreadPerBlock; i++) {
        final += sum[i];
    }
    
    pi = step * final;
    
    run_time = omp_get_wtime() - start_time;
    printf("\n pi with %ld steps is %lf in %lf seconds\n ",num_steps,pi,run_time);
}